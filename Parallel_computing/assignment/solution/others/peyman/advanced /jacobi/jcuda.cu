#include "hip/hip_runtime.h"
#include <unistd.h>
#include <stdlib.h>
#include <sys/types.h>
#include <sys/stat.h>
#include <fcntl.h>
#include <stdio.h>

#define N 4096

float *x, *y;
float *dev_x;
float *dev_y;
#define myabs(a) (((a) > 0) ? (a):(-(a)))

__global__ void matrixadd(float* x, float* y, int NN, float a1, float a2, float a3, float a4, float *diff)
{
  int  i = blockIdx.x * blockDim.x + threadIdx.x;  
  int  index = (NN+2)*(i+1);
  
  diff[i] = -1.0;
  for(int k=1; k <= NN; k++){
    x[index + k] = a2 * y[index + k - 1] + a4 * y[index + k + 1] + a1 * y[index + k - NN - 2] + a3 * y[index + k + NN + 2];
    if(myabs(x[index + k] - y[index + k]) > diff[i])
      diff[i] = myabs(x[index+k] - y[index+k]);
  }
  //__syncthreads();
}


int main(void) 
{
  int NN;
  float a1, a2, a3, a4, a5, a6;
  float MAXDIFF;
  int i, j;
  int t, t1, t2;
  float  maxdiff1;
  float *mydiff, *dev_mydiff;
  int iteration;
  FILE *fp;

  if((fp = fopen("input.jacobi","r+")) == NULL){
	printf("File not found.\n");
	exit(1);
}
  
  fscanf(fp,"%d %f %f %f %f %f %f %f", &NN, &a1, &a2, &a3, &a4, &a5, &a6, &MAXDIFF);
  

  printf("%d %f %f %f %f %f %f %f\n", NN, a1, a2, a3, a4, a5, a6, MAXDIFF);   
  /*  a1 = a2 = a3 = a4 = 0.25; a6 = 0; a5 = 0.1;

  MAXDIFF = 0.0001;
  */

  /* 1. allocate host memory */
  x = (float*)malloc( (NN+2)*(NN+2)*sizeof(float) );
  y = (float*)malloc( (NN+2)*(NN+2)*sizeof(float) );


  printf("maxdiff = %13.12f\n", MAXDIFF);

  for (i=1; i<=NN+1; i++) {
    x[i] = a5*i;
    y[i] = a5*i;

    x[i*(NN+2)] = 0.0;
    y[i*(NN+2)] = 0.0;

    x[i+(NN+1)*(NN+2)] = a6*i;
    y[i+(NN+1)*(NN+2)] = a6*i;

    x[NN+1+i*(NN+2)] = 0.0;
    y[NN+1+i*(NN+2)] = 0.0;
  }
  
  for (i=1; i<=NN; i++) 
    for (j=1; j<=NN; j++) {
      x[i+j*NN] = 0.0;
      y[i+j*NN] = 0.0;
    }

  printf("maxdiff=%13.12f\n", MAXDIFF);
  
  t = 0; t1 = 1;
  maxdiff1 = 100000.0;
  iteration = 0;
  mydiff = (float*) malloc( NN*sizeof(float) );
  
  int blockSize = 16;
 
  hipMalloc( &dev_x, (NN+2)*(NN+2)*sizeof(float) );
  hipMalloc( &dev_y, (NN+2)*(NN+2)*sizeof(float) ); 
  hipMalloc( &dev_mydiff, NN*sizeof(float) );

  hipMemcpy( dev_x, x, (NN+2)*(NN+2)*sizeof(float), hipMemcpyHostToDevice );
  hipMemcpy( dev_y, y, (NN+2)*(NN+2)*sizeof(float), hipMemcpyHostToDevice );

  while (maxdiff1 > MAXDIFF) {
    maxdiff1 = -1.0;
   
      
  hipMemcpy( dev_mydiff, mydiff, NN*sizeof(float), hipMemcpyHostToDevice );
   	       
   if (t == 0) matrixadd<<<(NN/blockSize), blockSize>>>(dev_x, dev_y, NN, a1, a2, a3, a4, dev_mydiff);
   else if(t == 1) matrixadd<<<(NN/blockSize), blockSize>>>(dev_y, dev_x, NN, a1, a2, a3, a4, dev_mydiff);

    hipMemcpy( mydiff, dev_mydiff, NN*sizeof(float), hipMemcpyDeviceToHost );
   
    for(i = 0; i < NN; i++){
      //for(j = 0; j < NN; j++){
	if(maxdiff1 < mydiff[i]) 
	  maxdiff1 = mydiff[i];
	mydiff[i] = -1;
	//}
    }

    t2 = t; t = t1; t1 = t2;
    printf("iteration = %d, maxdiff1 = %f, MAXDIFF = %f\n", 
           iteration++, maxdiff1, MAXDIFF);
  }
  printf("MAXDIFF = %f, maxdiff = %f\n", MAXDIFF, maxdiff1);
      hipMemcpy( x, dev_x, (NN+2)*(NN+2)*sizeof(float), hipMemcpyDeviceToHost );
  hipMemcpy( y, dev_y, (NN+2)*(NN+2)*sizeof(float), hipMemcpyDeviceToHost );  

  if ((fp = fopen("cuda.output", "w+")) < 0) {
    fprintf(stderr, "Cannot open file proj3.output.\n");
    exit(0);
  }
  if(t == 1){
  for (j = 0; j <= NN + 1; j++) {
    if ((t = fprintf(fp, "%.10f\t", x[NN/2+j*(NN+2)])) < 0) {
      fprintf(stderr, "write error %d %d.\n", j, t);
      exit(0);
    }
  }

  for (j = 0; j <= NN + 1; j++) {
    if ((t = fprintf(fp, "%.10f\t", x[j+(NN/2)*(NN+2)])) < 0) {
      fprintf(stderr, "write error. %d %d\n", j, t);
      exit(0);
    }
  }
}
else{
  for (j = 0; j <= NN + 1; j++) {
    if ((t = fprintf(fp, "%.10f\t", y[NN/2+j*(NN+2)])) < 0) {
      fprintf(stderr, "write error %d %d.\n", j, t);
      exit(0);
   }
  }

  for (j = 0; j <= NN + 1; j++) {
    if ((t = fprintf(fp, "%.10f\t", y[j+(NN/2)*(NN+2)])) < 0) {
      fprintf(stderr, "write error. %d %d\n", j, t);
      exit(0);
    }
  }
}
  close(i);  
  free(mydiff);
  free(x);
  free(y); 
  hipFree(dev_x);
  hipFree(dev_y);
  hipFree(dev_mydiff);
  return 0;
}
